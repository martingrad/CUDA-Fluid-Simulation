#include "hip/hip_runtime.h"
/*
 *
 *
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>          // CUDA FFT Libraries
#include <hip/hip_runtime_api.h>    // Helper functions for CUDA Error handling

// OpenGL Graphics includes
#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// FluidsGL CUDA kernel definitions
#include "CUDA-Fluid-Simulation-kernels.cuh"

typedef unsigned int  uint;
typedef unsigned char uchar;

hipArray *fluidData_velocity_GPU = 0;
hipArray *fluidData_pressure_GPU = 0;

texture<fluidPressureType, 3, hipReadModeNormalizedFloat> tex_pressure;         // 3D texture
texture<fluidVelocityType, 3, hipReadModeNormalizedFloat> tex_velocity;

/*
* initCuda
*/
extern "C"
void initCuda(void *fluidData_velocity, void* fluidData_pressure, hipExtent volumeSize)
{
	// Velocity data
	// create 3D array
	hipChannelFormatDesc channelDesc_v = hipCreateChannelDesc<fluidVelocityType>();
	checkCudaErrors(hipMalloc3DArray(&fluidData_velocity_GPU, &channelDesc_v, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms copyParams_v = { 0 };
	copyParams_v.srcPtr = make_hipPitchedPtr(fluidData_velocity, volumeSize.width*sizeof(fluidVelocityType), volumeSize.width, volumeSize.height);
	copyParams_v.dstArray = fluidData_velocity_GPU;
	copyParams_v.extent = volumeSize;
	copyParams_v.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams_v));

	// set texture parameters
	tex_velocity.normalized = true;                      // access with normalized texture coordinates
	tex_velocity.filterMode = hipFilterModeLinear;      // linear interpolation
	tex_velocity.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	tex_velocity.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex_velocity, fluidData_velocity_GPU, channelDesc_v));

	// Pressure data
	// create 3D array
	hipChannelFormatDesc channelDesc_p = hipCreateChannelDesc<fluidPressureType>();
	checkCudaErrors(hipMalloc3DArray(&fluidData_pressure_GPU, &channelDesc_p, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms copyParams_p = { 0 };
	copyParams_p.srcPtr = make_hipPitchedPtr(fluidData_pressure, volumeSize.width*sizeof(fluidPressureType), volumeSize.width, volumeSize.height);
	copyParams_p.dstArray = fluidData_pressure_GPU;
	copyParams_p.extent = volumeSize;
	copyParams_p.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams_p));

	// set texture parameters
	tex_pressure.normalized = true;                      // access with normalized texture coordinates
	tex_pressure.filterMode = hipFilterModeLinear;      // linear interpolation
	tex_pressure.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	tex_pressure.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex_pressure, fluidData_pressure_GPU, channelDesc_p));
}


/*
* Forward Euler
* x^n+1 = x^n + f(x^n,t^n)t
* The value of x at the next time step equals the current value of x plus the current rate of change,
* times the duration of the time step t
*/
__device__
void forwardEuler()
{

}

__global__
void advectVelocity_kernel(char *a, int *b)
{
	// voxel (i,j,k)
	//int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	//int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int k = (blockIdx.z * blockDim.z) + threadIdx.z;

	a[threadIdx.x] += b[threadIdx.x];
	
}

extern "C"
void advectVelocity()
{
	dim3 threadsPerBlock(pow(THREADS_PER_BLOCK, 1 / 3), pow(THREADS_PER_BLOCK, 1 / 3), pow(THREADS_PER_BLOCK, 1 / 3));
	dim3 numBlocks(VOLUME_SIZE_X / threadsPerBlock.x, VOLUME_SIZE_Y / threadsPerBlock.y, VOLUME_SIZE_Z / threadsPerBlock.z);
	//dim3 numBlocks(NUMBER_OF_BLOCKS ^ 1/3 /, NUMBER_OF_BLOCKS ^ 1 / 3, NUMBER_OF_BLOCKS ^ 1 / 3);

	//advectVelocity_GPU <<<numBlocks, threadsPerBlock>>>(1,2);

	const int N = 16;
	const int blocksize = 16;
	char a[N] = "Hello \0\0\0\0\0\0";
	int b[N] = { 15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

	char *ad;
	int *bd;
	const int csize = N*sizeof(char);
	const int isize = N*sizeof(int);

	printf("%s", a);

	hipMalloc((void**)&ad, csize);
	hipMalloc((void**)&bd, isize);
	hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

	dim3 dimBlock(blocksize, 1);
	dim3 dimGrid(1, 1);
	advectVelocity_kernel<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);

	printf("%s\n", a);
}

